
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2006-2018 Istituto Italiano di Tecnologia (IIT)
 * Copyright (C) 2007 Giacomo Spigler
 * All rights reserved.
 *
 * This software may be modified and distributed under the terms of the
 * BSD-3-Clause license. See the accompanying LICENSE file for details.
 */

extern "C" {

  __global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
    int i=0;

    for(i=threadIdx.x+blockIdx.x*blockDim.x; i<w*h; i+=blockDim.x*gridDim.x) {
      out[i*3]=in[i*3+2];
      out[i*3+1]=in[i*3+1];
      out[i*3+2]=in[i*3];
    }
  }

}

