
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2006-2018 Istituto Italiano di Tecnologia (IIT)
 * Copyright (C) 2007 Giacomo Spigler
 * All rights reserved.
 *
 * This software may be modified and distributed under the terms of the
 * BSD-3-Clause license. See the accompanying LICENSE file for details.
 */

extern "C" {

  __global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
    int i=0;

    for(i=threadIdx.x+blockIdx.x*blockDim.x; i<w*h; i+=blockDim.x*gridDim.x) {
      if(in[i*3]>=200) {
        //out[i*3]=255;
        //out[i*3+1]=255;
        //out[i*3+2]=255;
      } else {
        out[i*3]=0;
        out[i*3+1]=0;
        out[i*3+2]=0;
      }
    }
  }

}


